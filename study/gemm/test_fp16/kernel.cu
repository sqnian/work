#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <torch/extension.h>
#include "ixinfer.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <cassert>
#include <iostream>


template <typename T>
T prod(const T *arr, const size_t start, const size_t end) {
    T ret = 1;
    for (size_t i = start; i < end; i++) {
        ret *= arr[i];
    }
    return ret;
}

at::Tensor one_test(at::Tensor input, at::Tensor weight){

    TORCH_CHECK(input.scalar_type() == at::ScalarType::Half);
    TORCH_CHECK(weight.scalar_type() == at::ScalarType::Half);
    cuinferPointerMode_t cuinfer_ptr_mode = CUINFER_POINTER_MODE_HOST;
    cuinferOperation_t transa = CUINFER_OP_T;
    cuinferOperation_t transb = CUINFER_OP_N;
    hipDataType Atype = HIP_R_16F;
    hipDataType Btype = HIP_R_16F;
    hipDataType Ctype = HIP_R_16F;
    hipDataType computeType = HIP_R_32F;
    hipDataType scaleType = HIP_R_32F;
    cuinferGEMMCustomOption_t customOption;
    customOption = CUINFER_BLAS_GEMM_CUSTOM_NONE; 
    hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    cuinferHandle_t handle;
    cuinferCreate(&handle);
    float gemm_alpha = 1.0f;
    float gemm_beta = 0.f;

    at::Tensor output = input.new_empty({input.size(0), input.size(1),weight.size(0)});
    __half *res_ptr = (__half *)output.data_ptr();


    int m = weight.size(0);
    int n = input.size(1);
    int k = weight.size(1);

    std::cout << "m: " << m << " n: " << n << " k: " << k <<std::endl;
    int lda = k;
    int ldb = k;
    int ldc = m;  

    long strideA = 0;
    long strideB = 0;
    long strideC = 0;

    __half *wei_ptr = (__half *)weight.data_ptr();
    __half *inp_ptr = (__half *)input.data_ptr();

    int batch = 1;
    if (input.dim() > 2) {
        batch =  input.size(0);     // prod(input.data(), 0, input.dim() - 2);
    }
    std::cout << "batch: " << batch << std::endl;

    // if (input.dim() > 2) {
    //     strideB = input.size(1)* input.size(2);  // prod(inp_ptr, input.dim() - 2, input.dim());
    //     strideC = output.size(1)*output.size(2);   // prod(res_ptr, input.dim() - 2, input.dim());
    // }

    auto status =
        cuinferCustomGemm(handle, stream, cuinfer_ptr_mode, transa, transb,  m, n, k, &gemm_alpha, wei_ptr, Atype,
                          lda, strideA, inp_ptr, Btype, ldb, strideB, &gemm_beta, res_ptr, Ctype, ldc,
                          strideC, batch, computeType, scaleType, nullptr,  nullptr, customOption);
    if (status != CUINFER_STATUS_SUCCESS) {
        throw std::runtime_error("cuinferCustomGemm error!");
    }

    return output;

}